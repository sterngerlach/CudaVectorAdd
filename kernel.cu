#include "hip/hip_runtime.h"

/* kernel.cu */

#include <iostream>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <>

__global__ void AddVector(
    int vecSize, const float* vecA, const float* vecB, float* vecC)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < vecSize)
        vecC[i] = vecA[i] + vecB[i];
}

int main(int argc, char** argv)
{
    const int vecSize = 16384;
    hipError_t cudaErr = hipError_t::hipSuccess;

    std::cout << "vector addition of " << vecSize << " elements\n";

    /* �x�N�g���p�̃������̈���m�� */
    float* hostVecA = new (std::nothrow) float[vecSize];

    if (hostVecA == nullptr) {
        std::cerr << "failed to allocate sufficient memory for vector A\n";
        goto Cleanup;
    }

    float* hostVecB = new (std::nothrow) float[vecSize];

    if (hostVecB == nullptr) {
        std::cerr << "failed to allocate sufficient memory for vector B\n";
        goto Cleanup;
    }

    float* hostVecC = new (std::nothrow) float[vecSize];

    if (hostVecC == nullptr) {
        std::cerr << "failed to allocate sufficient memory for vector C\n";
        goto Cleanup;
    }

    /* �x�N�g��A��B�������� */
    for (int i = 0; i < vecSize; ++i) {
        hostVecA[i] = std::rand() / static_cast<float>(RAND_MAX);
        hostVecB[i] = std::rand() / static_cast<float>(RAND_MAX);
    }

    std::cout << "vector A and B initialized\n";

    /* �f�o�C�X�̃x�N�g���p�̃������̈���m�� */
    float* deviceVecA = nullptr;
    cudaErr = ::hipMalloc(&deviceVecA, vecSize * sizeof(float));

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to allocate device vector A: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    float* deviceVecB = nullptr;
    cudaErr = ::hipMalloc(&deviceVecB, vecSize * sizeof(float));

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to allocate device vector B: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    float* deviceVecC = nullptr;
    cudaErr = ::hipMalloc(&deviceVecC, vecSize * sizeof(float));

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to allocate device vector C: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    /* �x�N�g��A��B���z�X�g����f�o�C�X�ɓ]�� */
    cudaErr = ::hipMemcpy(deviceVecA, hostVecA, vecSize * sizeof(float), hipMemcpyHostToDevice);

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to copy vector A from host to device: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    cudaErr = ::hipMemcpy(deviceVecB, hostVecB, vecSize * sizeof(float), hipMemcpyHostToDevice);

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to copy vector B from host to device: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    /* �f�o�C�X��Ńx�N�g���̉��Z�����s */
    unsigned int threadsPerBlock = 256;
    unsigned int blocksPerGrid = (vecSize + threadsPerBlock - 1) / threadsPerBlock;
    
    dim3 dimGrid { blocksPerGrid, 1, 1 };
    dim3 dimBlock { threadsPerBlock, 1, 1 };
    
    std::cout << "launching CUDA kernel with " << blocksPerGrid
              << " blocks of " << threadsPerBlock << " threads\n";

    AddVector<<<dimGrid, dimBlock>>>(vecSize, deviceVecA, deviceVecB, deviceVecC);
    cudaErr = ::hipGetLastError();

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to launch AddVector kernel: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    /* �v�Z���ʂ��f�o�C�X����z�X�g�ɓ]�� */
    cudaErr = ::hipMemcpy(hostVecC, deviceVecC, vecSize * sizeof(float),
                           hipMemcpyDeviceToHost);

    if (cudaErr != hipError_t::hipSuccess) {
        std::cerr << "failed to copy vector C from device to host: "
                  << ::hipGetErrorString(cudaErr) << '\n';
        goto Cleanup;
    }

    /* �v�Z���ʂ̌��� */
    for (int i = 0; i < vecSize; ++i) {
        if (std::fabs(hostVecA[i] + hostVecB[i] - hostVecC[i]) > 1e-5) {
            std::cerr << "result verification failed at element "
                      << i << '\n';
            goto Cleanup;
        }
    }

    std::cout << "vector addition succeeded\n";

Cleanup:
    /* �f�o�C�X�̃x�N�g���p�̃������̈����� */
    if (deviceVecA != nullptr) {
        cudaErr = ::hipFree(deviceVecA);

        if (cudaErr != hipError_t::hipSuccess)
            std::cerr << "failed to free device vector A: "
                      << ::hipGetErrorString(cudaErr) << '\n';
    }

    if (deviceVecB != nullptr) {
        cudaErr = ::hipFree(deviceVecB);

        if (cudaErr != hipError_t::hipSuccess)
            std::cerr << "failed to free device vector B: "
                      << ::hipGetErrorString(cudaErr) << '\n';
    }

    if (deviceVecC != nullptr) {
        cudaErr = ::hipFree(deviceVecC);

        if (cudaErr != hipError_t::hipSuccess)
            std::cerr << "failed to free device vector C: "
                      << ::hipGetErrorString(cudaErr) << '\n';
    }

    /* �x�N�g���p�̃������̈����� */
    if (hostVecA != nullptr)
        delete[] hostVecA;

    if (hostVecB != nullptr)
        delete[] hostVecB;

    if (hostVecC != nullptr)
        delete[] hostVecC;

    return EXIT_SUCCESS;
}
